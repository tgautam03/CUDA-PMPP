
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

__global__ void histogram_parallel_shared(char *sentence_data_device, unsigned int sentence_len, unsigned int *histogram_device) 
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; // Which character this thread will work on

    __shared__ unsigned int sh_histogram[7]; // Allocating shared memory
    for (unsigned bin = threadIdx.x; bin < 7; bin+=blockDim.x)
    {
        sh_histogram[bin] = 0; // Initializing shared memory values to zero
    }
    __syncthreads(); // Barrier sync to ensure all values are initialized

    if (i < sentence_len)
    {
        int alphabet_pos = sentence_data_device[i] - 'a'; // Position of the alphabet
        if (alphabet_pos >= 0 && alphabet_pos < 26) // Store if it is lowercase
            atomicAdd(&sh_histogram[alphabet_pos/4], 1); // Storing in shared memory histogram
    }

    // Commit to global memory
    __syncthreads(); // Ensuring all threads in a block are done computing shared memory histograms
    // Assigning each thread to a bin in the shared memory histogram
    for (unsigned int bin = threadIdx.x; bin < 7; bin+=blockDim.x)
    {
        unsigned int bin_value = sh_histogram[bin]; // Getting the value in the bin
        if (bin_value > 0) // Checking if the value is non-zero
        {
            atomicAdd(&histogram_device[bin], bin_value); // Updating the value in the global histogram
        }
    }
}

int main(int argc, char const *argv[])
{
    std::string sentence; // Input string
    std::cout << "Enter the sentence (all lowercase): ";
    std::getline(std::cin, sentence); // Input from terminal stored in the variable sentence

    // Book keeping
    const char *sentence_data_host = sentence.c_str();
    size_t sentence_len = sentence.length() + 1; // +1 for null terminator

    unsigned int num_threads_per_block = 256; // Num threads per block
    unsigned int num_blocks = ceil(sentence_len/(float)num_threads_per_block); // Total number of blocks

    // Computing histogram
    unsigned int *histogram_host = new unsigned int[7];  // Histogram array with 7 bins to store 1) a-d, e-h, i-l, and so on.
    for (int i = 0; i < 7; i++)
        histogram_host[i] = 0;
    
    // Move sentence to GPU
    char *sentence_data_device;
    hipMalloc((void**)&sentence_data_device, sentence_len*sizeof(char));
    hipMemcpy(sentence_data_device, sentence_data_host, sentence_len*sizeof(char), hipMemcpyHostToDevice);

    // Move histogram to GPU
    unsigned int *histogram_device;
    hipMalloc((void**)&histogram_device, 7*sizeof(unsigned int));
    hipMemcpy(histogram_device, histogram_host, 7*sizeof(unsigned int), hipMemcpyHostToDevice);

    // For recording time
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    
    // Kernel execution
    hipEventRecord(beg);
    histogram_parallel_shared<<<num_blocks, num_threads_per_block>>>(sentence_data_device, sentence_len, histogram_device);
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time = elapsed_time * 1000.;

    std::cout << "Runtime : " << elapsed_time << " microseconds \n";

    // Move histogram result to CPU
    hipMemcpy(histogram_host, histogram_device, 7*sizeof(unsigned int), hipMemcpyDeviceToHost);

    // Display results
    std::cout << "\n--- Character Frequency Histogram ---\n";
    std::cout << "-----------------------------------\n";

    // Define the labels for each bin
    const std::string labels[] = {"a-d", "e-h", "i-l", "m-p", "q-t", "u-x", "y-z"};

    for (int i = 0; i < 7; ++i)
    {
        // Print each label and its corresponding count on a new line
        std::cout << "Range " << labels[i] << ": " << histogram_host[i] << std::endl;
    }
    std::cout << "-----------------------------------\n";

    return 0;
}
